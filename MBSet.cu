#include "hip/hip_runtime.h"
/*
 * File:   MBSet.cu
 *
 * Created on November 25, 2015
 *
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 *
 * Name: Amit Kulkarni
 * GTID: 903038158

 */
#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"
#include <GL/freeglut.h>

// Size of window in pixels, both width and height
#define WINDOW_DIM            512

using namespace std;

// Initial screen coordinates, both host and device.
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);

//device copy of iterations array
int *deviceIter;

// To test Swap
int Ga = 10;
int Gb = 20;

const int maxIt = 2000; // Msximum Iterations

// Define the RGB Class
class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};

RGB* colors = 0; // Array of color values

void InitializeColors()
{
  colors = new RGB[maxIt + 1];
  for (int i = 0; i < maxIt; ++i)
    {
      if (i < 5)
        { // Try this.. just white for small it counts
          colors[i] = RGB(1, 1, 1);
        }
      else
        {
          colors[i] = RGB(drand48(), drand48(), drand48());
        }
    }
  colors[maxIt] = RGB(); // black
}
// Function written in the Host but run in the device.
// 1] Calculate x & y index for every thread
// 2] iterate over all values (nIter < 2000)
// 3] copy all valid values to deviceIter
__global__ void calculateMBSet(int* deviceIter, double xD, double yD, double CminR, double CminI)
{

  // Step 1:
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int xIndex = index / WINDOW_DIM;
  int yIndex = index % WINDOW_DIM;

  Complex zo(0.0, 0.0);
  Complex c((CminR + xD * xIndex), (CminI + yD * yIndex));
  Complex z((CminR + xD * xIndex), (CminI + yD * yIndex));
  int it = 0;

  // Step 2:
  for(int i = 0; i < maxIt ; i++)
  {
    if(z.magnitude2() > 4.0) break;
      zo = z*z + c;
      z = zo;
      it++;
  }

  // Step 3:
  deviceIter[xIndex + yIndex * WINDOW_DIM] = it;

}

//Total size of iterations array
int winSize = sizeof(int) * WINDOW_DIM * WINDOW_DIM;

//Total result array of iterations
int *final = (int*) malloc(winSize);

void computeResult()
{
  // 1] Copy x & y values and calculate number of blocks required
  // 2] Create dynamic memory in Device memory (hipMalloc)
  // 3] Call __global__ function which runs in the Device
  // 4] Copy the calculated values back into the Host memory (hipMemcpy)
  // cout << "APPLE" << endl;
  // Step 1:
  double CminR = minC.r;
  double CminI = minC.i;
  double realD = maxC.r - minC.r;
  double imgD = maxC.i - minC.i;
  double xD = realD / WINDOW_DIM;
  double yD = imgD / WINDOW_DIM;
  int nBlocks = WINDOW_DIM * WINDOW_DIM/32;

  // Step 2:
  hipMalloc((void **)&deviceIter, winSize);

  // Step 3:
  calculateMBSet <<< nBlocks,32 >>> (deviceIter, xD, yD, CminR, CminI);

  // Step 4:
  hipMemcpy(final, deviceIter, winSize, hipMemcpyDeviceToHost);
  //cout << "computeResult called " << endl;

}

// 1] Get value from final
// 2] Get random color value and plot
void plotMBSetPixel()
{
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

  glBegin(GL_POINTS);
    for (int col = 0; col < WINDOW_DIM; col++)
    {
      for (int row = 0; row < WINDOW_DIM; row++)
      {
          // Step 1:
          int val = final[col + row * WINDOW_DIM];
          // Step 2:
          glColor3f(colors[val].r, colors[val].g, colors[val].b);
          glVertex2f(col, row);
      }
    }
  glEnd();

}

// Function to plot a sqaure using 4 given values
void plotSquare(int &a, int &b, int &c, int &d)
{

  glClear(GL_DEPTH_BUFFER_BIT);
  glLoadIdentity();
  gluOrtho2D(0, WINDOW_DIM, WINDOW_DIM, 0);
  glColor3f(1.0, 0.0, 0.0);

  glBegin(GL_LINE_LOOP);
    glVertex2f(a,b);
    glVertex2f(c,b);
    glVertex2f(c,d);
    glVertex2f(a,d);
  glEnd();

  glFlush();
  glutSwapBuffers();
}

// Function to swap to values
void SwapValue (int &a, int &b)
{
  int temp;
  temp = a;
  a = b;
  b = temp;

  return;
}

// Global stack declaration to maintain a history
stack<double> realCStack;
stack<double> imgCStack;

// 1] If 'q' is pressed, quit
// 2] If 'b' is pressed, check if both stacks arent empty.
// 3] Copy real and img values of maxC and minC from the stack
void keyboard (unsigned char key, int x, int y)
{
  // Use switch
  // Step 1:
  if(key == 'q')
  {
    exit(0);
  }

  // Step 2:
  if(key == 'b')
  {

    if( (!imgCStack.empty()) && (!realCStack.empty()))
    {
        // Step 3:
        maxC.i = imgCStack.top();
        imgCStack.pop();
        maxC.r = realCStack.top();
        realCStack.pop();
        minC.i = imgCStack.top();
        imgCStack.pop();
        minC.r = realCStack.top();
        realCStack.pop();
    }
    else
    {
      //cout << "INITIAL POSTION. CANNOT GO BACK!" << endl;
    }
    computeResult();
    glutPostRedisplay();
  }

}

// Global variables for mouse and zooming operation
bool flag = false; // check mouse button
int rMin, rMax, iMin, iMax;
int xStart, xEnd, yStart, yEnd; // For mouse operation

// 1] Get co-ordinate values when mouse is clicked. Min for button =  down. Max for button = up
// 2] Calculate factors if flag == true
// 3] Swap based on location
// 4] Reassign new values
// 5] Plot sqaure for zooming and recalculate. Reset flag for mouse click
void mouse(int button, int state, int x, int y)
{
  // Step 1:
  if(button == GLUT_LEFT_BUTTON && state == GLUT_UP)
  {
      rMax = x;
      iMax = y;
      flag = true;
  }

  if(button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
  {
      rMin = x;
      iMin = y;
      realCStack.push(minC.r);
      imgCStack.push(minC.i);
      realCStack.push(maxC.r);
      imgCStack.push(maxC.i);
  }

  if(flag)
  {
    // Step 2:
    xStart = rMin;
    yStart = iMin;
    xEnd = rMax;
    yEnd = iMax;

    double rDiff = maxC.r - minC.r;
    double rFactor = rDiff/(WINDOW_DIM-1);

    double iDiff = maxC.i - minC.i;
    double iFactor = iDiff/(WINDOW_DIM-1);

    int xDiff = xEnd - xStart;
    int yDiff = yEnd - yStart;

    // Step 3:
    if((yStart < yEnd) && (xStart < xEnd))
    {   // Right down. Don't swap
    }
    else if((yStart > yEnd) && (xStart > xEnd))
    {
        // up left
        // Swap both xStart and yStart
        SwapValue(xStart, xEnd);
        SwapValue(yStart, yEnd);
    }
    else if ((yStart < yEnd) && (xStart > xEnd))
    {   // down left
        // Swap xStart
        SwapValue(xStart, xEnd);
    }
    else if ((yStart > yEnd) && (xStart < xEnd))
    {
        // up right
        // swap yStart
        SwapValue(yStart, yEnd);
    }

    if(xDiff > yDiff)
    {
        xEnd = xStart + yDiff;
    }
    else if(xDiff < yDiff)
    {
        yEnd = yStart + xDiff;
    }

    // Step 4: Reassign values
    rMax = xEnd;
    rMin = xStart;
    iMax = yEnd;
    iMin = yStart;

    double rMinf = minC.r + (rFactor * rMin);
    double rMaxf = minC.r + (rFactor * rMax);
    double iMinf = minC.i + (iFactor * iMin);
    double iMaxf = minC.i + (iFactor * iMax);

    minC.r = rMinf;
    minC.i = iMinf;
    maxC.r = rMaxf;
    maxC.i = iMaxf;

    // Step 5:
    plotSquare(xStart, yStart, xEnd, yEnd);
    computeResult();
    flag = false;

    }
}

void display(void)
{
    glLoadIdentity();
    gluOrtho2D(0, WINDOW_DIM, WINDOW_DIM, 0);

    // Draw Mandelbrot
    plotMBSetPixel();

    // Swap the double buffers
    glutSwapBuffers();
}

void init(void)
{
  glEnable(GL_DEPTH_TEST);
  glShadeModel(GL_SMOOTH);
}

int main(int argc, char** argv)
{
  // Initialize OPENGL here
  // Set up necessary host and device buffers
  // set up the opengl callbacks for display, mouse and keyboard

  // Calculate the interation counts
  // Grad students, pick the colors for the 0 .. 1999 iteration count pixels
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
    glutInitWindowSize(WINDOW_DIM, WINDOW_DIM);
    glutInitWindowPosition(100, 100);
    glutCreateWindow("MandelBrot Set");
    glClearColor(1.0,1.0,1.0,1.0);
    computeResult();

    init();
    glMatrixMode(GL_MODELVIEW);
    InitializeColors();
    glutDisplayFunc(display);
    glutIdleFunc(display);
    glutKeyboardFunc (keyboard);
    glutMouseFunc(mouse);


  glutMainLoop(); // THis will callback the display, keyboard and mouse
  return 0;

}
